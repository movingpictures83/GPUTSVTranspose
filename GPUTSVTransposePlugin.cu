#include <emmintrin.h>
#include <sys/time.h> 
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

#include <iostream>
#include <iomanip>
#include <fstream>

#include "GPUTSVTransposePlugin.h"

void GPUTSVTransposePlugin::input(std::string file) {
 inputfile = file;
 std::ifstream ifile(inputfile.c_str(), std::ios::in);
 while (!ifile.eof()) {
   std::string key, value;
   ifile >> key;
   ifile >> value;
   parameters[key] = value;
 }
 N = atoi(parameters["N"].c_str());
 A = (double*) malloc(N*N*sizeof(double));
 C = (double*) malloc(N*N*sizeof(double));
 int M = N * N;
 std::ifstream myinput((std::string(PluginManager::prefix())+parameters["matrix"]).c_str(), std::ios::in);
 int i;
 for (i = 0; i < M; ++i) {
	int k;
	myinput >> k;
        A[i] = k;
 }
}




void GPUTSVTransposePlugin::run() {
	double *pA;
	double *pC;
hipMalloc((void**)&pA, (N*N)*sizeof(double));
hipMalloc((void**)&pC, (N*N)*sizeof(double));
hipMemcpy(pA, A, (N*N)*sizeof(double), hipMemcpyHostToDevice);
printf("***Transpose on %d x %d Matrix on GPU***\n",N,N);
MatTrans<<<N,N>>>(pA, pC, N);
hipMemcpy(C, pC, (N*N)*sizeof(double), hipMemcpyDeviceToHost);

hipFree(pA);
hipFree(pC);

}

void GPUTSVTransposePlugin::output(std::string file) {
	std::ofstream outfile(file.c_str(), std::ios::out);
        int i, j;
        for (i = 0; i < N; ++i){
            for (j = 0; j < N; ++j){
		outfile << C[i*N+j];//std::setprecision(0) << a[i*N+j];
		if (j != N-1)
			outfile << "\t";
		else
			outfile << "\n";
            }
	}
	free(A);
	free(C);
}



PluginProxy<GPUTSVTransposePlugin> GPUTSVTransposePluginProxy = PluginProxy<GPUTSVTransposePlugin>("GPUTSVTranspose", PluginManager::getInstance());


